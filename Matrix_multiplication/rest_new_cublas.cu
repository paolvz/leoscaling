#include <stdio.h>
#include <stdlib.h>
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <omp.h>


void check_result(double*C, double*B, int n_row, int n_col, int rank)
{
       int sum = 0;
            for (int i = 0; i < n_row; i++)
            {
                for (int j = 0; j < n_col; j++)
                {
                    if (C[i * n_col + j] == B[i * n_col + j])
                    {   
                        //printf("C[%d][%d] = %g, B[%d][%d] = %g\n", i, j, C[i * n_col + j], i, j, B[i * n_col + j]);
                        sum++;
                    }
                }
            }
            if (sum == n_row * n_col)
        {
            printf("Matrix multiplication is correct, rank: %d\n", rank);
        }
        else
        {
            printf("Matrix multiplication is incorrect, rank: %d\n", rank);
        }
        
}

void print_matrix(double*A, int n_row, int n_col)
{
    for (int i = 0; i < n_row; i++)
    {
        for (int j = 0; j < n_col; j++)
        {
            printf("%g ", A[i * n_col + j]);
        }
        printf("\n");
    }
    printf("\n");
}


    

int main(int argc, char **argv)
{     
    
    int N = MATRIX_SIZE;

    int rank, size;

     
    int thread_level;
    
    MPI_Init_thread(&argc, &argv, MPI_THREAD_FUNNELED, &thread_level);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);
    

     if (rank == 0){
    printf("GPU CUBLAS Matrix Multiplication\n");
    printf("Matrix Size: %d\n", N);
    }

    double max_comp_time;
    double max_comm_time;
    double max_init_time;
    double max_gpu_cpu_time;
    double final_comp = 0.0;
    double final_comm = 0.0;
    double final_init = 0.0;
    double final_gpu_cpu = 0.0;
    
    char name[MPI_MAX_PROCESSOR_NAME];
    int len;
    MPI_Get_processor_name( name, &len );

    
    

    /////// GPU /////////////
    int gpu_count;
    hipGetDeviceCount(&gpu_count);

    int gpunumber = rank % gpu_count;
    int device;
    hipSetDevice(gpunumber);
    hipGetDevice(&device);
    printf("Rank: %d, GPU: %d, Node: %s\n", rank, device, name);
    ///////////////////////////

    int N_LOC = N / size;
    int rest = N % size;
    int offset = 0;


    if (rank < rest) {N_LOC++;}else{offset = rest;}
 
    // allocate array with malloc
    double*A_LOC = (double*)calloc(N_LOC * N, sizeof(double));
    double*B_LOC = (double*)malloc(N * N_LOC * sizeof(double));
    
    

    // initialize A_LOC and B_LOC
    MPI_Barrier(MPI_COMM_WORLD);
    double start_init = MPI_Wtime();
    #pragma omp parallel for
    for (int i = 0; i < N_LOC; i++)
    {
        for (int j = 0; j < N; j++)
       
    {   if (j == i + N_LOC * rank +offset)
        {

        A_LOC[i*N+j] = 1;

        }
    }
    }

    #pragma omp parallel for
    for (int i = 0; i < N * N_LOC; i++)
    {
        B_LOC[i] = 10 + rank + 2;
    }
    
    double end_init = MPI_Wtime();
    final_init = end_init - start_init;
    

    

    // All Gather

    double* B_TEMP_N = (double*)malloc((long long int)(N / size + 1) * N * sizeof(double));
    double* B_TEMP = (double*)malloc((long long int)(N / size + 1) * N_LOC * sizeof(double));
    double*C_TEMP_N = (double*)malloc((long long int)N_LOC * N * sizeof(double));
    
    
    
    
    ////// GPU ///////
    double *cu_A_LOC;
    double *cu_B_TEMP_N;
    double *cu_C_TEMP_N;
    
    hipMalloc((void**)&cu_A_LOC, (long long int)N_LOC * N * sizeof(double));
    hipMalloc((void**)&cu_C_TEMP_N, (long long int)N_LOC * N * sizeof(double));
    hipMalloc((void**)&cu_B_TEMP_N, (long long int)(N / size + 1) * N * sizeof(double));



    double start_cpu_gpu;
    double end_cpu_gpu;

    start_cpu_gpu = MPI_Wtime();
    
    hipMemcpy(cu_A_LOC, A_LOC, N_LOC * N * sizeof(double), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    
    end_cpu_gpu = MPI_Wtime();
    final_gpu_cpu += end_cpu_gpu - start_cpu_gpu;
    
    double *cu_C_LOC = cu_C_TEMP_N;
    ////// GPU ///////

    int N_COL = N_LOC;
   
   

    


    for (int k = 0; k < size; k++)
    {
        
        MPI_Barrier(MPI_COMM_WORLD);
        double start_comm = MPI_Wtime();
        
        
        if (k < rest)
        {
            N_COL = N / size + 1;

            cu_C_LOC = cu_C_TEMP_N + (k * N_COL);
            
            

          
        }
        else
        {
            N_COL = N / size;
            
            cu_C_LOC = cu_C_TEMP_N + (k * N_COL + rest);
          
        }
        


        
        #pragma omp parallel for collapse(2)
        for (int i = 0; i < N_LOC; i++)
        {
            for (int j = 0; j < N_COL; j++)
            {

                if (k >= rest)
                {

                    B_TEMP[i * N_COL + j] = B_LOC[(k * N_COL + rest) + i * N + j];
                }
                else
                {

                    B_TEMP[i * N_COL + j] = B_LOC[(k * N_COL) + i * N + j];
                }
            }
        }

        int *receivecounts = (int *)malloc(size * sizeof(int));
        int *displs = (int *)malloc(size * sizeof(int));

        int sub_receivecounts = N_COL * N_LOC;
        MPI_Allgather(&sub_receivecounts, 1, MPI_INT, receivecounts, 1, MPI_INT, MPI_COMM_WORLD);

        displs[0] = 0;

        for (int i = 1; i < size; i++)
        {
            displs[i] = displs[i - 1] + receivecounts[i - 1];
        }
        

      
        MPI_Allgatherv(B_TEMP, N_COL * N_LOC, MPI_DOUBLE, B_TEMP_N, receivecounts, displs, MPI_DOUBLE, MPI_COMM_WORLD);
        
        double end_comm = MPI_Wtime();
        final_comm += end_comm - start_comm;

       
        
        
        /////////// GPU /////////////


        
        
        
        
        
        MPI_Barrier(MPI_COMM_WORLD);

        start_cpu_gpu = MPI_Wtime();

       
        hipMemcpy(cu_B_TEMP_N, B_TEMP_N, N_COL * N * sizeof(double), hipMemcpyHostToDevice);
        
        hipDeviceSynchronize();

        end_cpu_gpu = MPI_Wtime();
        final_gpu_cpu += end_cpu_gpu - start_cpu_gpu;


        
        const double alpha = 1.0;
        const double beta = 0.0;

        hipblasHandle_t handle;
        hipblasCreate(&handle);
        
    

        double start_comp = MPI_Wtime();

        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N_COL, N_LOC, N, &alpha, cu_B_TEMP_N, N_COL, cu_A_LOC, N, &beta, cu_C_LOC, N);
        hipDeviceSynchronize();
        
        
        double end_comp = MPI_Wtime();
        final_comp += end_comp - start_comp;


        
        hipblasDestroy(handle);
        

  
    }

    /////// GPU /////////////
    MPI_Barrier(MPI_COMM_WORLD);
    start_cpu_gpu = MPI_Wtime();

    hipMemcpy(C_TEMP_N, cu_C_TEMP_N, N_LOC * N * sizeof(double), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    end_cpu_gpu = MPI_Wtime();
    final_gpu_cpu += end_cpu_gpu - start_cpu_gpu;
    /////// GPU /////////////
    
    
    //check_result(C_TEMP_N, B_LOC, N, N_LOC, rank);

    #ifdef SAVE
    
        MPI_File file;
        MPI_Offset displacement;
        MPI_Status status;
        
        

        
        displacement = (rank * N_LOC  + offset )* N * sizeof(double);
        
        MPI_File_open(MPI_COMM_WORLD, "result.bin", MPI_MODE_CREATE | MPI_MODE_WRONLY, MPI_INFO_NULL, &file);
        MPI_File_set_view(file, displacement, MPI_CHAR, MPI_CHAR, "native", MPI_INFO_NULL);
        MPI_File_write(file, C_TEMP_N, N_LOC * N * sizeof(double), MPI_CHAR, &status);
        MPI_File_close(&file);
    
    #endif
    

    MPI_Reduce(&final_comp, &max_comp_time, 1, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD);
    MPI_Reduce(&final_comm, &max_comm_time, 1, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD);
    MPI_Reduce(&final_init, &max_init_time, 1, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD);
    MPI_Reduce(&final_gpu_cpu, &max_gpu_cpu_time, 1, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD);

   
      if (rank == 0)
        {
            printf("\nComputation Time (Max): %f seconds\n", max_comp_time);
            printf("Communication Time (Max): %f seconds\n", max_comm_time);
            printf("Initialization Time (Max): %f seconds\n", max_init_time);
            printf("GPU-CPU Time (Max): %f seconds\n", max_gpu_cpu_time);

            printf("\n%f %f %f %f %s\n", max_init_time, max_comm_time, max_comp_time, max_gpu_cpu_time, getenv("SLURM_NNODES"));
        }
    
    
    free(A_LOC);
    free(B_LOC);
    free(B_TEMP);
    free(B_TEMP_N);
    free(C_TEMP_N);

    hipFree(cu_A_LOC);
    hipFree(cu_B_TEMP_N);
    hipFree(cu_C_TEMP_N);
   
    




 

    MPI_Finalize();





    return 0;
}
